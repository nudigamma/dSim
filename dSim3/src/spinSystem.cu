#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////////
// File name:		spinSystem.cu
// Description:		Definition of all CUDA functions that are not used inside the
//			kernel.
/////////////////////////////////////////////////////////////////////////////////////////


#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include "spinKernel.cu"
#include "radixsort.cu"
//#include "dSimDataTypes.h"

extern "C"
{

void checkCUDA()
{
  gpuDeviceInit(0);
}


///////////////////////////////////////////////////////////////////////
// Function name:	allocateArray
// Description:		Allocate memory on device for an array pointed to
//			by devPtr of size size.
///////////////////////////////////////////////////////////////////////
void allocateArray(void **devPtr, size_t size)
{
	checkCudaErrors(hipMalloc(devPtr,size));
}


///////////////////////////////////////////////////////////////////////
// Function name:	freeArray
// Description:		Free up the device memory used by the array pointed
//			to by devPtr
///////////////////////////////////////////////////////////////////////
void freeArray(void *devPtr)
{
	checkCudaErrors(hipFree(devPtr));
}


///////////////////////////////////////////////////////////////////////
// Function name:	threadSync
// Description:		Block until the device has completed all preceding
//			requested tasks.
///////////////////////////////////////////////////////////////////////
void threadSync()
{
	checkCudaErrors(hipDeviceSynchronize());
}


///////////////////////////////////////////////////////////////////////
// Function name:	copyArrayFromDevice
// Description:		Copy array from device (pointed to by device parameter)
//			to array on host (pointed to by host parameter)
///////////////////////////////////////////////////////////////////////
void copyArrayFromDevice(void* host, const void* device, unsigned int vbo, int size)
{
	if (vbo)
		checkCudaErrors(cudaGLMapBufferObject((void**)&device, vbo));
	checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
	if (vbo)
		checkCudaErrors(cudaGLUnmapBufferObject(vbo));
}


////////////////////////////////////////////////////////////////////////
// Function name:	copyArrayToDevice
// Description:		Copy array from host (pointed to by host parameter)
//			to array on device (pointed to by device parameter)
////////////////////////////////////////////////////////////////////////
void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
	checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}


/////////////////////////////////////////////////////////////////////////
// Function name:	copyConstantToDevice
// Description:		Copy constant from host (with name host) to device
//			(with name device).
/////////////////////////////////////////////////////////////////////////
void copyConstantToDevice(void* device, const void* host, int offset, int size)
{
	checkCudaErrors(hipMemcpyToSymbol((char *) device, host, offset, size));
}


//////////////////////////////////////////////////////////////////////////
// Function name:	registerGLBufferObject
// Description:		Registers the buffer object of ID vbo for access by CUDA.
//////////////////////////////////////////////////////////////////////////
void registerGLBufferObject(uint vbo)
{
	checkCudaErrors(cudaGLRegisterBufferObject(vbo));
}


//////////////////////////////////////////////////////////////////////////
// Function name:	unregisterGLBufferObject
// Description:		Unregisters the buffer object of ID vbo for access by CUDA
//			and releases any CUDA resources associated with the buffer.
//////////////////////////////////////////////////////////////////////////
void unregisterGLBufferObject(uint vbo)
{
	checkCudaErrors(cudaGLUnregisterBufferObject(vbo));
}


//////////////////////////////////////////////////////////////////////////
// The following functions bind/unbind various arrays from host to device
// texture memory.
// Note: Should combine into one function
//////////////////////////////////////////////////////////////////////////
void bindCubeCounter(uint* ptr, int size)						// Test
{
	hipBindTexture(0,texCubeCounter,ptr,size*sizeof(uint));
}

void unbindCubeCounter()								// Test
{
	hipUnbindTexture(texCubeCounter);
}

void bindTrianglesInCubes(uint* ptr, int size)						// Test
{
	hipBindTexture(0,texTrianglesInCubes,ptr,size*sizeof(uint));
}

void unbindTrianglesInCubes()								// Test
{
	hipUnbindTexture(texTrianglesInCubes);
}
/*
void bindTrgls(uint* ptr, int size)							// Test
{
	hipBindTexture(0,texTrgls,ptr,size*sizeof(uint));
}

void unbindTrgls()									// Test
{
	hipUnbindTexture(texTrgls);
}
*/
void bindVertices(float* ptr, int size)							// Test
{
	if (size>0){
		hipBindTexture(0,texVertices,ptr,size*sizeof(float));
	}
}

void unbindVertices()									// Test
{
	hipUnbindTexture(texVertices);
}

void bindTriangleHelpers(float* ptr, int size)						// Test
{
	if (size>0){
		hipBindTexture(0,texTriangleHelpers,ptr,size*sizeof(float));
	}
}

void unbindTriangleHelpers()								// Test
{
	hipUnbindTexture(texTriangleHelpers);
}

void bindRTreeArray(float* ptr, int size)						// Test
{
	if (size>0){
		hipBindTexture(0,texRTreeArray,ptr,size*sizeof(float));
	}
}

void unbindRTreeArray()									// Test
{
	hipUnbindTexture(texRTreeArray);
}

void bindTreeIndexArray(uint* ptr, int size)						// Test
{
	if (size>0){
		hipBindTexture(0,texCombinedTreeIndex,ptr,size*sizeof(uint));
	}
}

void unbindTreeIndexArray()								// Test
{
	hipUnbindTexture(texCombinedTreeIndex);
}

void bindTriInfo(uint* ptr, int size)						// Test
{
	if (size>0){
		hipBindTexture(0,texTriInfo,ptr,size*sizeof(uint));
	}
}

void unbindTriInfo()								// Test
{
	hipUnbindTexture(texTriInfo);
}

///////////////////////////////////////////////////////////////////////////
// Function name:	integrateSystem
// Description:		Run the kernel for spin computations
///////////////////////////////////////////////////////////////////////////
void integrateSystem(
			float* pos,
			uint* randSeed,
			//float* spinInfo,
			spinData* spinInfo,
			float deltaTime,
			float permeability,
			int numBodies,
			float3 gradient,
			float phaseConstant,
			uint iterations, uint* trianglesInCubes, uint* cubeCounter
			)
{
	static bool firstCall = true;
	struct hipDeviceProp_t devInfo;
	hipGetDeviceProperties(&devInfo,0);

	if (firstCall){
		firstCall = false;
		// Write out some info
		printf("CUDA device info:\n\n");
		printf("Name: %s\n", devInfo.name);
		printf("totalGlobalMem: %u\n", devInfo.totalGlobalMem);
		printf("sharedMemPerBlock: %u\n", devInfo.sharedMemPerBlock);
		printf("regsPerBlock: %u\n", devInfo.regsPerBlock);
		printf("warpSize: %u\n", devInfo.warpSize);
		printf("memPitch: %u\n", devInfo.memPitch);
		printf("maxThreadsPerBlock: %u\n", devInfo.maxThreadsPerBlock);
		printf("\n\n");
	}

	// Number of threads will normally be 128
	int numThreads = min(128, numBodies);
	int numBlocks = 1 + numBodies/numThreads;
	
	// Execute the kernel
	integrate<<< numBlocks, numThreads >>>( 
						(float3*) pos,
						(uint2*) randSeed,
						//(float4*) spinInfo,
						spinInfo,
						deltaTime,
						permeability,
						numBodies,
						gradient.x, gradient.y, gradient.z,
						phaseConstant,
						iterations, trianglesInCubes, cubeCounter);

	getLastCudaError("Kernel execution failed\n");
}


//////////////////////////////////////////////////////////////////////////////////////
// Function name:	integrateSystemVBO
// Description:		Register the vertex buffer object for access by CUDA, perform
//			the GPU computation using integrateSystem, then unregister
//			the VBO.
//////////////////////////////////////////////////////////////////////////////////////
void integrateSystemVBO(
			uint vboPos,
			uint* randSeed,
			//float* spinInfo,
			spinData* spinInfo,
			float deltaTime,
			float permeability,
			int numBodies,
			float3 gradient,
			float phaseConstant,
			uint iterations, uint* trianglesInCubes, uint* cubeCounter
			)
{
	float *pos;
	checkCudaErrors(cudaGLMapBufferObject((void**)&pos, vboPos));
	integrateSystem(pos,randSeed,spinInfo,deltaTime,permeability, numBodies, gradient, phaseConstant, iterations, trianglesInCubes, cubeCounter);
	checkCudaErrors(cudaGLUnmapBufferObject(vboPos));
}


} // extern "C"
