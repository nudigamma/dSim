#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////////////
// File name:		spinKernel.cu
// Description:		Kernel for spin computations using GPU
///////////////////////////////////////////////////////////////////////////////////////

#ifndef _SPIN_KERNEL_H_
#define _SPIN_KERNEL_H_

#include <stdio.h>
#include <math.h>
#include <hip/hip_vector_types.h>
//#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "options.h"
#include "dSimDataTypes.h"

#define PI 3.14159265358979f
#define TWOPI 6.28318530717959f


//////////////////////////////////////////////////////////////////////////////////
// Define texture arrays and constants, copied to device from host.
//////////////////////////////////////////////////////////////////////////////////
texture<uint,1,hipReadModeElementType> texCubeCounter;
texture<uint,1,hipReadModeElementType> texTrianglesInCubes;
//texture<uint,1,hipReadModeElementType> texTrgls;
texture<float,1,hipReadModeElementType> texVertices;
texture<float,1,hipReadModeElementType> texTriangleHelpers;
texture<float,1,hipReadModeElementType> texRTreeArray;
texture<uint,1,hipReadModeElementType> texCombinedTreeIndex;
texture<uint,1,hipReadModeElementType> texTriInfo;

__constant__ uint k_reflectionType;
__constant__ uint k_triSearchMethod;
__constant__ uint k_numCubes;
__constant__ uint k_totalNumCubes;
__constant__ uint k_maxTrianglesPerCube;
__constant__ float k_cubeLength;
__constant__ uint k_nFibers; 
__constant__ uint k_nCompartments;
__constant__ float k_permeability;
__constant__ float k_deltaTime;
__constant__ float *k_T2Values;
__constant__ float *k_stdDevs;

typedef unsigned int uint;

/////////////////////////////////////////////////////////////////////////////////////
// The structure collResult will be used to store outcomes from checks of whether
// collision occurs between a ray and a triangle.
/////////////////////////////////////////////////////////////////////////////////////
typedef struct _collResult
{
	uint collisionType;			// 0 if no collision, 1 if collision within triangle, 2 if collision with triangle edge, 3 if collision with triangle vertex
	float3 collPoint;			// Point of collision with triangle
	uint collIndex;				// Index of collision triangle
	float collDistSq;			// Distance squared from starting point to collision point
}collResult;


// Some simple vector ops for float3's (dot and length are defined in cudautil_math)
//#define dot(u,v)   ((u).x * (v).x + (u).y * (v).y + (u).z * (v).z)
//#define length(v)    sqrt(dot(v,v))  // norm (vector length)
#define d(u,v)	length(u-v)	// distance (norm of difference)


//////////////////////////////////////////////////////////////////////////
// Function name:	point_line_dist
// Description:		Returns the shortest distance from a point P to a 
//			line defined by two points (LP1 and LP2)
//////////////////////////////////////////////////////////////////////////
__device__ float point_line_dist(float3 P, float3 LP1, float3 LP2){
	float3 v = LP2-LP1;
	float b = dot(P-LP1,v)/dot(v,v);
	return d(P,LP1+b*v);
}


///////////////////////////////////////////////////////////////////////////
// Function name:	point_seg_dist
// Description:		Returns the shortest distance from a point P to a 
//			line segment defined by two points (SP1 and SP2)
///////////////////////////////////////////////////////////////////////////
__device__ float point_seg_dist(float3 P, float3 SP1, float3 SP2){
	float3 v = SP2-SP1;
	float c1 = dot(P-SP1,v);
	if (c1<=0) return d(P,SP1);
	float c2 = dot(v,v);
	if (c2<=c1) return d(P,SP2);
	float3 Pb = SP1 + c1/c2*v;
	return d(P,Pb);
}


//////////////////////////////////////////////////////////////////////////////
// Function name:	boxMuller
// Description:		Generates a pair of independent standard normally
//			distributed random numbers from a pair of
//			uniformly distributed random numbers, using the basic form
//			of the Box-Muller transform 
//			(see http://en.wikipedia.org/wiki/Box%E2%80%93Muller_transform)
//////////////////////////////////////////////////////////////////////////////
__device__ void boxMuller(float& u1, float& u2){
	float r = sqrtf(-2.0f * __logf(u1));
	float phi = TWOPI * u2;
	u1 = r * __cosf(phi);
	u2 = r * __sinf(phi);
}


//////////////////////////////////////////////////////////////////////////////
// Function name:	myRand
// Description:		Simple multiply-with-carry PRNG that uses two seeds 
//			(seed[0] and seed[1]) (Algorithm from George Marsaglia: 
//			http://en.wikipedia.org/wiki/George_Marsaglia)
//////////////////////////////////////////////////////////////////////////////
//__device__ uint myRand(uint seed[]){
//	seed[0] = 36969 * (seed[0] & 65535) + (seed[0] >> 16);
//	seed[1] = 18000 * (seed[1] & 65535) + (seed[1] >> 16);
//	return (seed[0] << 16) + seed[1];
//}
__device__ uint myRand(uint2 &seed){
	seed.x = 36969 * (seed.x & 65535) + (seed.x >> 16);
	seed.y = 18000 * (seed.y & 65535) + (seed.y >> 16);
	return (seed.x << 16) + seed.y;
}


/////////////////////////////////////////////////////////////////////////////
// Function name:	myRandf
// Description:		Returns a random float r in the range 0<=r<=1
/////////////////////////////////////////////////////////////////////////////
//__device__ float myRandf(uint seed[]){
//	return ((float)myRand(seed) / 4294967295.0f);
//}


/////////////////////////////////////////////////////////////////////////////
// Function name:	myRandDir
// Description:		Return a vector with a specified magnitude (adc) and 
//			a random direction
/////////////////////////////////////////////////////////////////////////////
//__device__ void myRandDir(uint seed[], float adc, float3& vec){
//	// Azimuth and elevation are on the interval [0,2*pi]
//	// (2*pi)/4294967294.0 = 1.4629181e-09f
//	float az = (float)myRand(seed) * 1.4629181e-09f;
//	float el = (float)myRand(seed) * 1.4629181e-09f;
//	vec.z = adc * __sinf(el);
//	float rcosel = adc * __cosf(el);
//	vec.x = rcosel * __cosf(az);
//	vec.y = rcosel * __sinf(az);
//	return;
//}


//////////////////////////////////////////////////////////////////////////////
// Function name:	myRandn
// Description:		Returns three normally distributed random numbers 
//			and one uniformly distributed random number.
//////////////////////////////////////////////////////////////////////////////
/*__device__ void myRandn(uint seed[], float& n1, float& n2, float& n3, float& u){
	// We want random numbers in the range (0,1], i.e. 0<n<=1
	n1 = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
	n2 = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
	n3 = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
	u = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
	// Note that ULONG_MAX=4294967295
	float n4 = u;
	boxMuller(n1,n2);
	boxMuller(n3,n4);
	return;
}*/
__device__ void myRandn(uint2 &seed, float& n1, float& n2, float& n3, float& u){
	// We want random numbers in the range (0,1], i.e. 0<n<=1
	n1 = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
	n2 = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
	n3 = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
	u = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
	// Note that ULONG_MAX=4294967295
	float n4 = u;
	boxMuller(n1,n2);
	boxMuller(n3,n4);
	return;
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Function name:	calcCubePosGPU										// Rename later to calcCubePos(...)
// Description: 	Function calculates the cube cell to which the given position belongs in uniform cube.
//			Converts a position coordinate (ranging from (-1,-1,-1) to (1,1,1) to a cube
//			coordinate (ranging from (0,0,0) to (m_numCubes-1, m_numCubes-1, m_numCubes-1)).
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__ uint3 calcCubePosGPU(float3 p){
	uint3 cubePos;
	cubePos.x = floor((p.x + 1.0f) / k_cubeLength);
	cubePos.y = floor((p.y + 1.0f) / k_cubeLength);
	cubePos.z = floor((p.z + 1.0f) / k_cubeLength);

	cubePos.x = max(0, min(cubePos.x, k_numCubes-1));
	cubePos.y = max(0, min(cubePos.y, k_numCubes-1));
	cubePos.z = max(0, min(cubePos.z, k_numCubes-1));

	return cubePos;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Function name:	calcCubeHashGPU										// Rename later to calcCubeHash(...)
// Description:		Calculate address in cube from position (clamping to edges)
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__ uint calcCubeHashGPU(uint3 cubePos){							
	return cubePos.z * k_numCubes * k_numCubes + cubePos.y * k_numCubes + cubePos.x;
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Function name:	reflectPos
// Description:		Given a particle that tries to travel from startPos to targetPos, but collides with triangle
//			number collTriIndex at collPos, we calculate the position which the particle gets reflected to.
//				This applies if reflectionType==1. If reflectionType==0, we do a simplified reflection,
//			where the particle just gets reflected to its original position. This is also done if we hit
//			a triangle edge or a triangle vertex (which gives collisionType==2 or collisionTYpe==3).
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__ float3 reflectPos(float3 startPos, float3 targetPos, float3 collPos, uint collTriIndex, uint collisionType){

	float3 reflectedPos;

	if ((k_reflectionType==0)|(collisionType>1)){			// We simply reflect back to the starting point
			reflectedPos = startPos;
	} else {				// We reflect the target point through the triangle - see http://en.wikipedia.org/wiki/Transformation_matrix
			float3 sPosShifted = targetPos-collPos;
			float3 normalVec;
			normalVec = make_float3(tex1Dfetch(texTriangleHelpers,collTriIndex*12+0),tex1Dfetch(texTriangleHelpers,collTriIndex*12+1),tex1Dfetch(texTriangleHelpers,collTriIndex*12+2));
			reflectedPos.x = (1-2*normalVec.x*normalVec.x)*sPosShifted.x - 2*normalVec.x*normalVec.y*sPosShifted.y - 2*normalVec.x*normalVec.z*sPosShifted.z + collPos.x;
			reflectedPos.y = -2*normalVec.x*normalVec.y*sPosShifted.x + (1-2*normalVec.y*normalVec.y)*sPosShifted.y - 2*normalVec.y*normalVec.z*sPosShifted.z + collPos.y;
			reflectedPos.z = -2*normalVec.x*normalVec.z*sPosShifted.x - 2*normalVec.y*normalVec.z*sPosShifted.y + (1-2*normalVec.z*normalVec.z)*sPosShifted.z + collPos.z;
	}

	return reflectedPos;
}


//////////////////////////////////////////////////////////////////////////////////////////////
// Function name:	triCollDetect
// Description:		Find whether the path from oPos to pos intersects triangle no. triIndex.
// 			Returns the collision result, which consists of 
// 				result.collPoint = the collision/intersection point between 
//							the ray and the triangle.
// 				result.collIndex = the index of the collision triangle if 
//							collision occurs
// 				result.collisionType = 0 if no collision, 1 within triangle boundaries,
//							2 if collision with triangle edge, 3 if 
//							collision with triangle vertex
// 				result.collDistSq = the distance (squared) from oPos to 
//							the collision point.
//////////////////////////////////////////////////////////////////////////////////////////////
__device__ collResult triCollDetect(float3 oPos, float3 pos, uint triIndex){

	uint firstPointIndex;
	float uv, uu, vv, wu, wv, r, s, t, stDen;
	float3 triP1, d, w, n, u, v, collPoint;
	collResult result;
	result.collisionType = 0;
	
	// firstPointIndex is the index of the "first" point in the triangle
	firstPointIndex = tex1Dfetch(texTriInfo, triIndex*3+2);
	// triP1 holds the coordinates of the first point
	triP1 = make_float3(tex1Dfetch(texVertices,firstPointIndex*3+0),tex1Dfetch(texVertices,firstPointIndex*3+1),tex1Dfetch(texVertices,firstPointIndex*3+2));
	// n: normal to the triangle. u: vector from first point to second point. v: vector from first point to third point. uv, uu, vv: dot products.
	n = make_float3(tex1Dfetch(texTriangleHelpers,triIndex*12+0),tex1Dfetch(texTriangleHelpers,triIndex*12+1),tex1Dfetch(texTriangleHelpers,triIndex*12+2));
	u = make_float3(tex1Dfetch(texTriangleHelpers,triIndex*12+3),tex1Dfetch(texTriangleHelpers,triIndex*12+4),tex1Dfetch(texTriangleHelpers,triIndex*12+5));
	v = make_float3(tex1Dfetch(texTriangleHelpers,triIndex*12+6),tex1Dfetch(texTriangleHelpers,triIndex*12+7),tex1Dfetch(texTriangleHelpers,triIndex*12+8));

	uv = tex1Dfetch(texTriangleHelpers,triIndex*12+9);
	uu = tex1Dfetch(texTriangleHelpers,triIndex*12+10);
	vv = tex1Dfetch(texTriangleHelpers,triIndex*12+11);

	// First find whether the path intersects the plane defined by triangle i. See method at http://softsurfer.com/Archive/algorithm_0105/algorithm_0105.htm
	r = dot(n,triP1-oPos)/dot(n,pos-oPos);

	if ((0<r)&(r<1)){
	// Then find if the path intersects the triangle itself. See method at http://softsurfer.com/Archive/algorithm_0105/algorithm_0105.htm
		d = r*(pos-oPos);
		collPoint = oPos + d;
		w = collPoint-triP1;

		wu = dot(w,u);
		wv = dot(w,v);

		stDen = uv*uv-uu*vv;
		s = (uv*wv-vv*wu)/stDen;
		t = (uv*wu-uu*wv)/stDen;

		if ( (s>=0)&(t>=0)&(s+t<=1) ){	// We have a collision with the triangle

			result.collDistSq = dot(d,d);
			result.collIndex = triIndex;
			result.collPoint = collPoint;
			result.collisionType = 1;

			if ( (s==0)|(t==0)|(s+t==1) ){						// The collision point is on a triangle edge
				result.collisionType = 2;
						
				if ( ((s==0)&(t==0))|((s==0)&(t==1))|((s==1)&(t==0)) ){		// The collision point is on a triangle vertex
					result.collisionType = 3;
				}							
			}
		}
	}
	return result;
}


/////////////////////////////////////////////////////////////////////////////////////////
// Function name:	SearchRTreeArray
// Description:		Find the leaf rectangles in the R-Tree which intersect the rectangle
//			rect (=[x_min,y_min,z_min,x_max,y_max,z_max]). Normally, rect will
//			be a bounding rectangle for a particle path and the leaf rectangles
//			of the R-Tree will be bounding rectangles for fiber triangles.
//			When the rectangles intersect, that means the particle might collide
//			with the triangle. The indices of such triangles are written into
//			intersectArray (to be further checked for actual collisions), and
//			the number of intersecting rectangles is returned in the output
//			foundCount.
/////////////////////////////////////////////////////////////////////////////////////////
__device__ uint SearchRTreeArray(float* rect, uint* interSectArray, uint8 &compartment, uint16 &fiberInside){
	uint foundCount = 0;
	uint stack[100];		// Maximum necessary stack size should be 1+7*(treeHeight) = 1+7*(n_levels-1). 100 should suffice for n_levels <= 15 - very big tree	
	int stackIndex = 0;
	
	//printf("k_nFibers: %u\n", k_nFibers);
	//printf("k_nCompartments: %u\n", k_nCompartments);
	//uint k_nFibers = 17, k_nCompartments = 3;
	//stack[stackIndex] = 0;
	if (compartment != 0){										// We push the location of the root node onto the stack
		stack[stackIndex] = tex1Dfetch(texCombinedTreeIndex,fiberInside*(k_nCompartments-1)+compartment);	// = 0 for "first" tree, i.e. tree corresponding to innermost compartment
	} else{
		stack[stackIndex] = tex1Dfetch(texCombinedTreeIndex,0);
		//printf("k_nFibers: %u\n", k_nFibers);
		//printf("k_nCompartments: %u\n", k_nCompartments);
		//printf("StackIndex: %u\n", stackIndex);
		//printf("Stack for compartment %u: %i\n", compartment, stack[stackIndex]);
	}
	//printf("(in spinKernel.cu::SearchRTreeArray): rect: [%g,%g,%g,%g,%g,%g]\n", rect[0],rect[1],rect[2],rect[3],rect[4],rect[5]);
	//printf("(in spinKernel.cu::SearchRTreeArray): stack[%i]: %u\n", stackIndex, stack[stackIndex]);
	stackIndex++;

	uint currentNodeIndex;
	


	while (stackIndex > 0){					// Stop when we've emptied the stack
		stackIndex--;					// Pop the top node off the stack
		currentNodeIndex = stack[stackIndex];
		//printf("(in spinKernel.cu::SearchRTreeArray): currentNodeIndex: %u\n", currentNodeIndex);

		for (int m=tex1Dfetch(texRTreeArray,currentNodeIndex+1)-1; m>=0; m--){
			uint currentBranchIndex = currentNodeIndex+2 + m*7;
			//printf("(in spinKernel.cu::SearchRTreeArray): m: %u\n", m);
			//printf("(in spinKernel.cu::SearchRTreeArray): currentBranchIndex: %u\n", currentBranchIndex);

			//See if the branch rectangle overlaps with the input rectangle
			if (!(  tex1Dfetch(texRTreeArray,currentBranchIndex+1) > rect[3] ||		// branchRect.x_min > rect.x_max
				tex1Dfetch(texRTreeArray,currentBranchIndex+2) > rect[4] ||		// branchRect.y_min > rect.y_max
				tex1Dfetch(texRTreeArray,currentBranchIndex+3) > rect[5] ||		// branchRect.z_min > rect.z_max
				rect[0] > tex1Dfetch(texRTreeArray,currentBranchIndex+4) ||		// rect.x_min > branchRect.x_max
				rect[1] > tex1Dfetch(texRTreeArray,currentBranchIndex+5) ||		// rect.y_min > branchRect.y_max
				rect[2] > tex1Dfetch(texRTreeArray,currentBranchIndex+6) ))		// rect.z_min > branchRect.z_max
			{	
				if (tex1Dfetch(texRTreeArray,currentNodeIndex) > 0){		// We are at an internal node - push the node pointed to in the branch onto the stack
					stack[stackIndex] = tex1Dfetch(texRTreeArray,currentBranchIndex);
					stackIndex++;
					//printf("(in spinKernel.cu::SearchRTreeArray): stackIndex: %i\n", stackIndex);
				} else {
					interSectArray[foundCount] = tex1Dfetch(texRTreeArray,currentBranchIndex); // We are at a leaf - store corresponding triangle index
					foundCount++;
					//printf("(in spinKernel.cu::SearchRTreeArray): Tree rectangle: [%g,%g,%g,%g,%g,%g]\n", tex1Dfetch(texRTreeArray,currentBranchIndex+1), tex1Dfetch(texRTreeArray,currentBranchIndex+2),
					//tex1Dfetch(texRTreeArray,currentBranchIndex+3), tex1Dfetch(texRTreeArray,currentBranchIndex+4), tex1Dfetch(texRTreeArray,currentBranchIndex+5),
					//tex1Dfetch(texRTreeArray,currentBranchIndex+6));
				}
			}
		}
	}
	return foundCount;
}


//////////////////////////////////////////////////////////////////////////////////////////
// Function name:	collDetectRTree
// Description:		See whether a particle trying to go from startPos to targetPos
//			collides with any triangle in the mesh, using the R-Tree. Return
//			the final position of the particle.
//////////////////////////////////////////////////////////////////////////////////////////
__device__ float3 collDetectRTree(float3 startPos, float3 targetPos, float u, uint8 &compartment, uint16 &fiberInside){
	
	float3 endPos = targetPos;
	uint hitArray[1200];				// Hitarray will store the indices of the triangles that the particle possible collides with - we are assuming no more than 100
	float spinRectangle[6];
	collResult result, tempResult;
	//float minCollDistSq;
	result.collDistSq = 400000000;			// Some really large number, will use this to store the smallest distance to a collision point
	result.collisionType = 1;
	result.collIndex = UINT_MAX;
	uint excludedTriangle = UINT_MAX;
	float u_max = 1, u_min = 0;
	//uint k = 0;
	//uint p = 0;

	//printf("Compartment: %i\n", compartment);

	while (result.collisionType>0){			// If we have detected a collision, we repeat the collision detection for the new, reflected path
		//minCollDistSq = 400000000;
		//printf("p: %u\n", p);
		//p++;
		result.collisionType = 0;		// First assume that the particle path does not experience any collisions

		// Define a rectangle that bounds the particle path from corner to corner
		// Finding minx, miny, minz
		spinRectangle[0] = startPos.x; if (targetPos.x < spinRectangle[0]){spinRectangle[0] = targetPos.x;}
		spinRectangle[1] = startPos.y; if (targetPos.y < spinRectangle[1]){spinRectangle[1] = targetPos.y;}
		spinRectangle[2] = startPos.z; if (targetPos.z < spinRectangle[2]){spinRectangle[2] = targetPos.z;}
	
		// Finding maxx, maxy, maxz
		spinRectangle[3] = startPos.x; if (targetPos.x > spinRectangle[3]){spinRectangle[3] = targetPos.x;}
		spinRectangle[4] = startPos.y; if (targetPos.y > spinRectangle[4]){spinRectangle[4] = targetPos.y;}
		spinRectangle[5] = startPos.z; if (targetPos.z > spinRectangle[5]){spinRectangle[5] = targetPos.z;}
	
		// Find the triangles whose bounding rectangles intersect spinRectangle. They are written to hitArray and their number is nHits.
		int nHits = SearchRTreeArray(spinRectangle, hitArray, compartment, fiberInside);
		//int nHits = 0;
		
		//printf("(in spinKernel.cu::collDetectRTree): nHits: %i\n", nHits);
		//printf("(in spinKernel.cu::collDetectRTree): Startpos: [%g,%g,%g]\n", startPos.x, startPos.y, startPos.z);
		//printf("(in spinKernel.cu::collDetectRTree): Targetpos: [%g,%g,%g]\n", targetPos.x, targetPos.y, targetPos.z);
		//printf("(in spinKernel.cu::collDetectRTree): Compartment: %i\n", compartment);
		//printf("(in spinKernel.cu::collDetectRTree): Fiber: %u\n", fiberInside);
		//printf("(in spinKernel.cu::collDetectRTree): Excluded triangle: %u\n", excludedTriangle);
		//printf("(in spinKernel.cu::collDetectRTree): result.collDistSq: %g\n", result.collDistSq);
	
		// Loop through the triangles in hitArray, see if we have collisions, store the closest collision point in the variable result.
		for (uint k=0; k<nHits; k++){
			uint triIndex = hitArray[k];
			//printf("(in spinKernel.cu::collDetectRTree): hitArray[%u]: %u\n", k, hitArray[k]);
			if (triIndex != excludedTriangle){
				tempResult = triCollDetect(startPos, targetPos, triIndex);
				//if ((tempResult.collisionType>0) & (tempResult.collDistSq < result.collDistSq)){
				if ((tempResult.collisionType>0) & (tempResult.collDistSq < result.collDistSq)){
					result = tempResult;
					//minCollDistSq = tempResult.collDistSq;
				}
			}
		}
		
	
		// If we have a collision, then we find the resulting point which the particle gets reflected to.
		if (result.collisionType>0){
			//printf("*\n");
			//printf("(in spinKernel.cu::collDetectRTree): Collision!\n");
			//printf("(in spinKernel.cu::collDetectRTree): startPos: [%g,%g,%g]\n", startPos.x,startPos.y,startPos.z);
			//printf("(in spinKernel.cu::collDetectRTree): targetPos: [%g,%g,%g]\n", targetPos.x,targetPos.y,targetPos.z);
			//printf("(in spinKernel.cu::collDetectRTree): Collision point: [%g,%g,%g]\n", result.collPoint.x, result.collPoint.y, result.collPoint.z);
			//printf("(in spinKernel.cu::collDetectRTree): Endpos (before assignment): [%g,%g,%g]\n", endPos.x, endPos.y, endPos.z);
			//printf("(in spinKernel.cu::collDetectRTree): Collision triangle index: %u\n", result.collIndex);
			//printf("(in spinKernel.cu::collDetectRTree): Collision fiber index: %u\n", tex1Dfetch(texTriInfo, result.collIndex*3+0));
			//printf("(in spinKernel.cu::collDetectRTree): Collision membrane index: %u\n", tex1Dfetch(texTriInfo, result.collIndex*3+1));
			//printf("(in spinKernel.cu::collDetectRTree): u: %g\n", u);
			//printf("(in spinKernel.cu::collDetectRTree): u_max: %g, u_min: %g, u_p: %g\n", u_max, u_min, u_max-(u_max-u_min)*k_permeability);

			// If u>u_max-(u_max-u_min)*k_permeability, then the particle permeates through the membrane and does not get reflected.
			// u is in the range (0,1].
			if (u<=u_max-(u_max-u_min)*k_permeability){		// The spin does not permeate the membrane
				endPos = reflectPos(startPos, targetPos, result.collPoint, result.collIndex, result.collisionType);
				u_max = u_max-(u_max-u_min)*k_permeability;
				//printf("(in spinKernel.cu::collDetectRTree): Particle bounces off membrane\n");
				//printf("(in spinKernel.cu::collDetectRTree): Endpos: [%g,%g,%g]\n", endPos.x, endPos.y, endPos.z);
				//reflectPos(startPos, targetPos, result.collPoint, result.collIndex, result.collisionType);
			} else{							// The spin permeates the membrane
				u_min = u_max-(u_max-u_min)*k_permeability;

				// Change the compartment (and fiber, if appropriate) assignment of the spin
				// uint membraneType = tex1Dfetch(texTriInfo, result.collIndex*3+1);
				if (compartment == 2){
					if (tex1Dfetch(texTriInfo, result.collIndex*3+1) == 0){		// We are going from compartment 2 through axon surface - new compartment is 1
						compartment = 1;
					} else {							// We are going from compartment 2 through myelin surface - new compartment is 0
						compartment = 0;
						fiberInside = UINT16_MAX;
					}
				} else if (compartment == 1){
					compartment = 2;						// We are going from compartment 1 through axon surface - new compartment is 2
				} else if (compartment == 3){
					compartment = 0;						// We are going from compartment 3 through glia surface - new compartment is 0
					fiberInside = UINT16_MAX;
				} else {
					fiberInside = tex1Dfetch(texTriInfo, result.collIndex*3+0);
					if (tex1Dfetch(texTriInfo, result.collIndex*3+1) == 1){		// We are going from compartment 0 through myelin surface - new compartment is 2
						compartment = 2;
					} else {							// We are going from compartment 0 through glia surface - new compartment is 3
						compartment = 3;
					}
				}
				
				//printf("(in spinKernel.cu::collDetectRTree): Particle permeates membrane\n");
				//printf("(in spinKernel.cu::collDetectRTree): Endpos: [%g,%g,%g]\n", endPos.x, endPos.y, endPos.z);
			}
		}

		// Redefine the start and end points for the reflected path, then repeat until no collision is detected.
		startPos = result.collPoint;
		targetPos = endPos;
		excludedTriangle = result.collIndex;					// Make sure we don't detect a collision with the triangle which the particle bounces from
		result.collDistSq = 400000000;
	}

	return endPos;
}


/////////////////////////////////////////////////////////////////////////////////////////////
// Function name:	cubeCollDetect
// Description:		Determine whether a particle traveling from oPos to pos experiences
//			a collision with any of the triangles in cube no. cubeIndex. Triangle
//			no. excludedTriangle is not checked - useful if the particle is bouncing
//			off that triangle.
/////////////////////////////////////////////////////////////////////////////////////////////
__device__ collResult cubeCollDetect(float3 oPos, float3 pos, uint cubeIndex, uint excludedTriangle, uint* trianglesInCubes, uint* cubeCounter){

	uint triIndex, k_max;
	collResult result, testCollision;
	result.collisionType = 0;
	result.collDistSq = 400000000;
	result.collIndex = UINT_MAX;

	// Loop through membrane types (layers) as appropriate
	//for (uint layerIndex = 0; layerIndex < 2; layerIndex++){					// Change later so not to loop through all membrane types
		//k_max = tex1Dfetch(texCubeCounter, layerIndex*k_totalNumCubes+cubeIndex);		// k_max: the number of triangles in cube cubeIndex on membrane type layerIndex
		//k_max = tex1Dfetch(texCubeCounter, cubeIndex);
		//cubeIndex = 1275;
		k_max = cubeCounter[cubeIndex];
		//printf("cubeCounter[%u]: %u\n", cubeIndex, k_max);
		for (uint k=0; k<k_max; k++){
			// triIndex is the number of the triangle being checked.
			//triIndex = tex1Dfetch(texTrianglesInCubes, (layerIndex*k_totalNumCubes+cubeIndex)*k_maxTrianglesPerCube+k);
//			triIndex = tex1Dfetch(texTrianglesInCubes, cubeIndex*k_maxTrianglesPerCube+k);
			triIndex = trianglesInCubes[cubeIndex*k_maxTrianglesPerCube+k];
			//printf("Checking triangle %u\n", triIndex);
			if (triIndex != excludedTriangle){
				testCollision = triCollDetect(oPos, pos, triIndex);

				if ( (testCollision.collisionType>0)&(testCollision.collDistSq<result.collDistSq) ){
					result = testCollision;
				}
			}
		}
		//triIndex = tex1Dfetch(texTrianglesInCubes, cubeIndex*k_maxTrianglesPerCube+k);
	//}

	return result;
}



///////////////////////////////////////////////////////////////////////////////////////////////
// Function name:	collDetectRectGrid
// Description:		Determine whether a particle trying to go from startPos to targetPos
//			collides with a triangle, using the method of a rectangular grid (as 
//			opposed	to an R-Tree)
///////////////////////////////////////////////////////////////////////////////////////////////
__device__ float3 collDetectRectGrid(float3 startPos, float3 targetPos, float u, uint8 compartment, uint16 fiberInside, uint* trianglesInCubes, uint* cubeCounter){

	float3 endPos = targetPos;
	collResult collCheck;
	collCheck.collisionType = 1;
	uint excludedTriangle = UINT_MAX, currCube;
	uint3 currCubexyz, startCubexyz, endCubexyz;
	int3 cubeIncrement;
	float u_max = 1.0f, u_min = 0.0f;

	while (collCheck.collisionType > 0){

		//startCube = calcCubeHashGPU(calcCubePosGPU(startPos, k_cubeLength), k_numCubes);		// The cube that the particle starts in
		//endCube = calcCubeHashGPU(calcCubePosGPU(targetPos, k_cubeLength), k_numCubes);			// The cube that the particle tries to end in
		
		startCubexyz = calcCubePosGPU(startPos);
		endCubexyz = calcCubePosGPU(targetPos);
		cubeIncrement.x = ( (endCubexyz.x>startCubexyz.x) - (endCubexyz.x<startCubexyz.x) );
		cubeIncrement.y = ( (endCubexyz.y>startCubexyz.y) - (endCubexyz.y<startCubexyz.y) );
		cubeIncrement.z = ( (endCubexyz.z>startCubexyz.z) - (endCubexyz.z<startCubexyz.z) );

		//printf("startCubexyz: [%u,%u,%u]\n", startCubexyz.x, startCubexyz.y, startCubexyz.z);
		//printf("endCubexyz: [%u,%u,%u]\n", endCubexyz.x, endCubexyz.y, endCubexyz.z);
		//printf("cubeIncrement: [%i,%i,%i]\n", cubeIncrement.x, cubeIncrement.y, cubeIncrement.z);

		collCheck.collisionType = 0;

		currCubexyz.x = startCubexyz.x;
		do {
			currCubexyz.y = startCubexyz.y;
			do {
				currCubexyz.z = startCubexyz.z;
				do {
					currCube = calcCubeHashGPU(currCubexyz);
					//printf("currCubexyz: [%u,%u,%u]\n", currCubexyz.x, currCubexyz.y, currCubexyz.z);
					collCheck = cubeCollDetect(startPos, targetPos, currCube, excludedTriangle, trianglesInCubes, cubeCounter);
					currCubexyz.z += cubeIncrement.z;
				} while ((currCubexyz.z != endCubexyz.z+cubeIncrement.z)&&(collCheck.collisionType == 0));
				currCubexyz.y += cubeIncrement.y;
			} while ((currCubexyz.y != endCubexyz.y+cubeIncrement.y)&&(collCheck.collisionType == 0));
			currCubexyz.x += cubeIncrement.x;
		} while ((currCubexyz.x != endCubexyz.x+cubeIncrement.x)&&(collCheck.collisionType == 0));



		/*while ((currCubexyz.x != endCubexyz.x+cubeIncrement.x)&&(collCheck.collisionType == 0)){
			while ((currCubexyz.y != endCubexyz.y+cubeIncrement.y)&&(collCheck.collisionType == 0)){
				while ((currCubexyz.z != endCubexyz.z+cubeIncrement.z)&&(collCheck.collisionType == 0)){
					currCubexyz.z += cubeIncrement.z;
					currCube = calcCubeHashGPU(currCubexyz);
					printf("currCubexyz: [%u,%u,%u]\n", currCubexyz.x, currCubexyz.y, currCubexyz.z);
					collCheck = cubeCollDetect(startPos, targetPos, currCube, excludedTriangle, trianglesInCubes, cubeCounter);
				}
				currCubexyz.y += cubeIncrement.y;
			}
			currCubexyz.x += cubeIncrement.x;
		}*/



		if (collCheck.collisionType > 0){

			//printf("(in collDetectRectGrid): Collision!\n");
			//printf("(in collDetectRectGrid): Startpos: [%g,%g,%g]\n", startPos.x, startPos.y, startPos.z);
			//printf("(in collDetectRectGrid): Targetpos: [%g,%g,%g]\n", targetPos.x, targetPos.y, targetPos.z);
			//printf("(in collDetectRectGrid): Collision pos: [%g,%g,%g]\n", collCheck.collPoint.x, collCheck.collPoint.y, collCheck.collPoint.z);
			//printf("(in collDetectRectGrid): Collision triangle: %u\n", collCheck.collIndex);
			//printf("(in collDetectRectGrid): Cube: %u\n", currCube);
			//printf("(in collDetectRectGrid): Compartment: %u\n", compartment);
			//printf("(in collDetectRectGrid): FiberInside: %u\n", fiberInside);
			
			if (u<=u_max-(u_max-u_min)*k_permeability){		// The spin does not permeate the membrane
				endPos = reflectPos(startPos, targetPos, collCheck.collPoint, collCheck.collIndex, collCheck.collisionType);
				u_max = u_max-(u_max-u_min)*k_permeability;
				//printf("(in spinKernel.cu::collDetectRTree): Particle bounces off membrane\n");
				//printf("(in spinKernel.cu::collDetectRTree): Endpos: [%g,%g,%g]\n", endPos.x, endPos.y, endPos.z);
				//reflectPos(startPos, targetPos, collCheck.collPoint, collCheck.collIndex, collCheck.collisionType);
			} else{							// The spin permeates the membrane
				u_min = u_max-(u_max-u_min)*k_permeability;

				// Change the compartment (and fiber, if appropriate) assignment of the spin
				// uint membraneType = tex1Dfetch(texTriInfo, collCheck.collIndex*3+1);
				if (compartment == 2){
					if (tex1Dfetch(texTriInfo, collCheck.collIndex*3+1) == 0){		// We are going from compartment 2 through axon surface - new compartment is 1
						compartment = 1;
					} else {							// We are going from compartment 2 through myelin surface - new compartment is 0
						compartment = 0;
						fiberInside = UINT16_MAX;
					}
				} else if (compartment == 1){
					compartment = 2;						// We are going from compartment 1 through axon surface - new compartment is 2
				} else if (compartment == 3){
					compartment = 0;						// We are going from compartment 3 through glia surface - new compartment is 0
					fiberInside = UINT16_MAX;
				} else {
					fiberInside = tex1Dfetch(texTriInfo, collCheck.collIndex*3+0);
					if (tex1Dfetch(texTriInfo, collCheck.collIndex*3+1) == 1){		// We are going from compartment 0 through myelin surface - new compartment is 2
						compartment = 2;
					} else {							// We are going from compartment 0 through glia surface - new compartment is 3
						compartment = 3;
					}
				}
			}
		}

		
		// Redefine the start and end points for the reflected path, then repeat until no collision is detected.
		startPos = collCheck.collPoint;
		targetPos = endPos;
		excludedTriangle = collCheck.collIndex;					// Make sure we don't detect a collision with the triangle which the particle bounces from
	}
	return endPos;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////
// Function name:	collDetect
// Description:		Determine whether a particle trying to travel from oPos to pos hits a triangle.
//			Use either the method of a rectangular grid or an R-Tree.
////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__ float3 collDetect(float3 oPos, float3 pos, float u, uint8 &compartment, uint16 &fiberInside, uint* trianglesInCubes, uint* cubeCounter){

	//if (k_triSearchMethod == 0){
		return collDetectRectGrid(oPos,pos,u,compartment,fiberInside,trianglesInCubes,cubeCounter);
	//} else {
	//	return collDetectRTree(oPos, pos, u, compartment, fiberInside);
	//}
	//return pos;
}



///////////////////////////////////////////////////////////////////////////////////////////////////////
// Function name:	integrate
// Description:		"Main" function for GPU kernel computation, called from spinSystem.cu, invokes all
//			the functions above. Computes the spin movement and signal for each spin by
//			performing the below computation in parallel on multiple threads.
///////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void integrate(
				float3* oldPos,
				uint2* oldSeed,
				//float4* spinInfo,
				spinData* spinInfo,
				float deltaTime,
				float permeability,
				int numBodies,
				float gradX, float gradY, float gradZ,
				float phaseConstant,
				uint iterations, uint* trianglesInCubes, uint* cubeCounter){

	int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index>=numBodies)
	return;

	float3 pos = oldPos[index];								// pos = particle position
	uint2 seed2 = oldSeed[index];								// seed4 = seed values (currently only using first 2 values)
	//float signalMagnitude = spinInfo[index].signalMagnitude;
	//float signalPhase = spinInfo[index].signalPhase;
	uint8 compartment = spinInfo[index].compartmentType;
	uint16 fiberInside = spinInfo[index].insideFiber;
	

/////////////////////////////////////////////////////////////////////////////////
// Now apply the brownian motion (free diffusion). We simulate brownian motion
// with a random walk where the x, y, and z componenets are drawn from a 
// normal distribution with mean 0 and standard deviation of sqrt(2*ADC*deltaTime).
// From wikipedia http://en.wikipedia.org/wiki/Random_walk:
//    In 3D, the variance corresponding to the Green's function of the diffusion equation is:
//       sigma^2 = 6*D*t
//    sigma^2 corresponds to the distribution associated to the vector R that links the two 
//    ends of the random walk, in 3D. The variance associated to each component Rx, Ry or Rz 
//    is only one third of this value (still in 3D).
// Thus, the standard deviation of each component is sqrt(2*ADC*deltaTime)
//////////////////////////////////////////////////////////////////////////////////

	//uint rseed[2];
	//rseed[0] = seed2.x;
	//rseed[1] = seed2.y;

	for (uint i=0; i<iterations; i++){

		// Take a random walk...
		// myRandn returns 3 PRNs from a normal distribution with mean 0 and SD of 1. 
		// So, we just need to scale these with the desired SD to get the displacements
		// for the random walk.
		// myRandn also returns a bonus uniformly distributed PRN as a side-effect of the 
		// Box-Muller transform used to generate normally distributed PRNs.
		float u;
		float3 brnMot;
		//myRandn(rseed, brnMot.y, brnMot.x, brnMot.z, u);
		myRandn(seed2, brnMot.y, brnMot.x, brnMot.z, u);
		float3 oPos = pos;						// Store a copy of the old position before we update it

		pos.x += brnMot.x * k_stdDevs[compartment];
		pos.y += brnMot.y * k_stdDevs[compartment];
		pos.z += brnMot.z * k_stdDevs[compartment];

		


		// Test
		if (index == 0){
			//printf("i = %u\n", i);
			//printf("index: %u\n", index);
			//printf("oPos: [%g,%g,%g]\n", oPos.x,oPos.y,oPos.z);
			//printf("pos: [%g,%g,%g]\n", pos.x,pos.y,pos.z);
			//printf("Compartment: %u\n", compartment);
			//printf("Fiberinside: %u\n", fiberInside);
			//printf("Signal magnitude: %g\n", signalMagnitude);
			//printf("Signal phase: %g\n", signalPhase);
			//printf("u (before assignment): %g\n", u);
			
			//printf("rseed after: [%u,%u]\n", rseed[0], rseed[1]);
			//printf("[%g,%g,%g,%g,%g,%g,%u,%u]\n", oPos.x, oPos.y, oPos.z, pos.x, pos.y, pos.z, compartment, fiberInside);

		
			//oPos.x = 0.0; oPos.y = 0.0; oPos.z = 0.01;		// oPos.x = 0.7; oPos.y = 0.0; oPos.z = 0.01;
			//pos.x = 0.1; pos.y = 0.2; pos.z = -0.01;		// pos.x = 0.632; pos.y = 0.067; pos.z = 0.01;
			//compartment = 1;
			//fiberInside = 0;
			//u = 0.9;
			//printf("u (after assignment): %g\n", u);
		}

		// Do a collision detection for the path the particle is trying to take
		pos = collDetect(oPos,pos,u,compartment,fiberInside,trianglesInCubes,cubeCounter);

		
		// Don't let the spin leave the volume
		if (pos.x > 1.0f)  { pos.x = 1.0f; /*signalMagnitude = 0.0;*/ }
		else if (pos.x < -1.0f) { pos.x = -1.0f; /*signalMagnitude = 0.0;*/ }
		if (pos.y > 1.0f)  { pos.y = 1.0f; /*signalMagnitude = 0.0;*/ }
		else if (pos.y < -1.0f) { pos.y = -1.0f; /*signalMagnitude = 0.0;*/ }
		if (pos.z > 1.0f)  { pos.z = 1.0f; /*signalMagnitude = 0.0;*/ }
		else if (pos.z < -1.0f) { pos.z = -1.0f; /*signalMagnitude = 0.0;*/ }

		// Update MR signal magnitude
		//signalMagnitude += -signalMagnitude/k_T2Values[compartment]*k_deltaTime;
		spinInfo[index].signalMagnitude += -spinInfo[index].signalMagnitude/k_T2Values[compartment]*k_deltaTime;
		
		// Update MR signal phase
		//signalPhase += (gradX * pos.x + gradY * pos.y + gradZ * pos.z) * phaseConstant;
		spinInfo[index].signalPhase += (gradX * pos.x + gradY * pos.y + gradZ * pos.z) * phaseConstant;

	}

	// Store new position
	//oldPos[index] = make_float4(pos, signalPhase);
	oldPos[index] = pos;

	// Store new seed values
	//oldSeed[index].x = rseed[0];
	//oldSeed[index].y = rseed[1];
	oldSeed[index].x = seed2.x;
	oldSeed[index].y = seed2.y;

	// Store new values of compartment and signal magnitude and phase
	//spinInfo[index].signalMagnitude = signalMagnitude;
	//spinInfo[index].signalPhase = signalPhase;
	spinInfo[index].compartmentType = compartment;
	spinInfo[index].insideFiber = fiberInside;
		
}

#endif
